#include<stdio.h>
#include<hip/hip_runtime.h>


#define N 512
#define BLOCK_SIZE 16
 
__global__ void MatAdd(float *A, float *B, float *C){
    int i =blockIdx.x * blockDim.x + threadIdx.x;
    int j =blockIdx.y * blockDim.y + threadIdx.y;
    
    if(i<N && j<N)
        C[i*N+j]=A[i*N+j]+B[i*N+j];
}

int main(){
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int i;

    h_A = (float*)malloc(N*N*sizeof(float));
    h_B = (float*)malloc(N*N*sizeof(float));
    h_C = (float*)malloc(N*N*sizeof(float));

    //init data
    for(i=0;i<(N*N);i++){
        h_A[i]=1.0;
        h_B[i]=2.0;
        h_C[i]=0.0;
    }

    //allocate device memory
    hipMalloc((void**)&d_A, N*N*sizeof(float));
    hipMalloc((void**)&d_B, N*N*sizeof(float));
    hipMalloc((void**)&d_C, N*N*sizeof(float));

    //transfe data to device

    hipMemcpy(d_A,h_A,N*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,N*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_C,h_C,N*N*sizeof(float),hipMemcpyHostToDevice);

    dim3 blockSize(1,1);
    dim3 numBlock(N,N);

    MatAdd<<<numBlock,blockSize>>>(d_A,d_B,d_C);
    hipDeviceSynchronize();

    hipMemcpy(h_C,d_C,N*N*sizeof(float),hipMemcpyDeviceToHost);
/*
    for(i<0;i<N*N;i++){
        if(h_C[i]!=3.0)
            printf("ERRORR:%f,idx:%d\n",h_C[i],i);
            break;

    }*/
    printf("PASS!!!!!!!!!!!!!!!\n");

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
